#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <chrono>

using namespace cv;
using namespace std;
using namespace std::chrono;

#define HEIGHT 512 // must be multiple of block_size.y
#define WIDTH 512 // must be multiple of block_size.x
#define MAX_ITER 10000

void draw_mandelbrot_gpu();
void mandelbrotGPU(char*);
__global__ void calc(char* image_buffer);

#define cudaAssertSuccess(ans) { _cudaAssertSuccess((ans), __FILE__, __LINE__); }
inline void _cudaAssertSuccess(hipError_t code, char* file, int line)
{
    if (code != hipSuccess) {
        fprintf(stderr, "_cudaAssertSuccess: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

//int main(int argc, char** argv)
void draw_mandelbrot_gpu()
{
    Mat image_output(HEIGHT, WIDTH, CV_8UC1);
    char* image_data = new char[HEIGHT * WIDTH];
    memcpy(image_data, image_output.data, HEIGHT * WIDTH);

    // start time measurement
    auto start_time = high_resolution_clock::now();

    mandelbrotGPU(image_data);

    // stop time measurement
    auto stop_time = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop_time - start_time);
    //cout << "Time taken: " << duration.count() << " milliseconds" << endl;

    Mat image_output_show(HEIGHT, WIDTH, CV_8UC1, image_data);

    // show duration time
    if (WIDTH > 199) {
        putText(image_output_show, "Time taken: " + to_string(duration.count()) + "ms", Point(10, 20), FONT_HERSHEY_SIMPLEX, 0.5, Scalar(255, 255, 255), 2);
    }

    imshow("GPU", image_output_show);
    waitKey(0);
    delete[] image_data;

}

void mandelbrotGPU(char* image_buffer)
{
    char* d_image_buffer;
    cudaAssertSuccess(hipMalloc(&d_image_buffer, WIDTH * HEIGHT));
    dim3 block_size(16, 16);
    dim3 grid_size(WIDTH / block_size.x, HEIGHT / block_size.y);
    calc << <grid_size, block_size >> > (d_image_buffer);
    cudaAssertSuccess(hipPeekAtLastError());
    cudaAssertSuccess(hipDeviceSynchronize());
    cudaAssertSuccess(hipMemcpy(image_buffer, d_image_buffer, HEIGHT * WIDTH, hipMemcpyDeviceToHost));
    cudaAssertSuccess(hipFree(d_image_buffer));
}

__global__ void calc(char* image_buffer)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // WIDTH
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // HEIGHT
    int idx = row * WIDTH + col;
    if (col >= WIDTH || row >= HEIGHT) return;

    float x0 = ((float)col / WIDTH) * 3.5f - 2.5f;
    float y0 = ((float)row / HEIGHT) * 3.5f - 1.75f;

    float x = 0.0f;
    float y = 0.0f;
    int iter = 0;
    float xtemp;
    while ((x * x + y * y <= 4.0f) && (iter < MAX_ITER))
    {
        xtemp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = xtemp;
        iter++;
    }

    int color = iter * 5;
    if (color >= 256) color = 0;
    image_buffer[idx] = color;
}

int main(int argc, char** argv)
{
    draw_mandelbrot_gpu();
}

